#include "hip/hip_runtime.h"
#include "../include/matrix.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

// Kernel básico para multiplicación elemento por elemento
__global__ void elementwise_add_kernel(float* a, float* b, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] + b[idx];
    }
}

// Kernel básico para multiplicación por escalar
__global__ void scalar_multiply_kernel(float* data, float scalar, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = data[idx] * scalar;
    }
}

// Kernel básico para multiplicación de matrices (versión simple)
__global__ void matrix_multiply_kernel(float* a, float* b, float* c, 
                                     int rows_a, int cols_a, int cols_b) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows_a && col < cols_b) {
        float sum = 0.0f;
        for (int k = 0; k < cols_a; ++k) {
            sum += a[row * cols_a + k] * b[k * cols_b + col];
        }
        c[row * cols_b + col] = sum;
    }
}

// Función para suma con CUDA
Matrix Matrix::cuda_add(const Matrix& other) const {
    // Verificar dimensiones
    if (rows != other.rows || cols != other.cols) {
        throw std::runtime_error("Dimensiones incompatibles para suma CUDA");
    }
    
    int size = rows * cols;
    size_t bytes = size * sizeof(float);
    
    // Alocar memoria en GPU
    float *d_a, *d_b, *d_result;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_result, bytes);
    
    // Copiar datos a GPU
    hipMemcpy(d_a, data.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, other.data.data(), bytes, hipMemcpyHostToDevice);
    
    // Configurar grid y bloques
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    
    // Ejecutar kernel
    elementwise_add_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_result, size);
    
    // Verificar errores
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Error CUDA: " << hipGetErrorString(err) << std::endl;
    }
    
    // Crear matriz resultado y copiar datos de vuelta
    Matrix result(rows, cols);
    hipMemcpy(result.data.data(), d_result, bytes, hipMemcpyDeviceToHost);
    
    // Liberar memoria GPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
    
    return result;
}

// Función para multiplicación por escalar con CUDA
Matrix Matrix::cuda_scalar_multiply(float scalar) const {
    int size = rows * cols;
    size_t bytes = size * sizeof(float);
    
    float *d_data, *d_result;
    hipMalloc(&d_data, bytes);
    hipMalloc(&d_result, bytes);
    
    hipMemcpy(d_data, data.data(), bytes, hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    
    scalar_multiply_kernel<<<gridSize, blockSize>>>(d_data, scalar, d_result, size);
    
    Matrix result(rows, cols);
    hipMemcpy(result.data.data(), d_result, bytes, hipMemcpyDeviceToHost);
    
    hipFree(d_data);
    hipFree(d_result);
    
    return result;
}

// Función para multiplicación de matrices con CUDA
Matrix Matrix::cuda_multiply(const Matrix& other) const {
    if (cols != other.rows) {
        throw std::runtime_error("Dimensiones incompatibles para multiplicación CUDA");
    }
    
    size_t bytes_a = rows * cols * sizeof(float);
    size_t bytes_b = other.rows * other.cols * sizeof(float);
    size_t bytes_c = rows * other.cols * sizeof(float);
    
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_b, bytes_b);
    hipMalloc(&d_c, bytes_c);
    
    hipMemcpy(d_a, data.data(), bytes_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, other.data.data(), bytes_b, hipMemcpyHostToDevice);
    
    // Configurar grid 2D para multiplicación de matrices
    dim3 blockSize(16, 16);
    dim3 gridSize((other.cols + blockSize.x - 1) / blockSize.x,
                  (rows + blockSize.y - 1) / blockSize.y);
    
    matrix_multiply_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, 
                                                   rows, cols, other.cols);
    
    Matrix result(rows, other.cols);
    hipMemcpy(result.data.data(), d_c, bytes_c, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return result;
}

// Función optimizada con cuBLAS para multiplicación de matrices
Matrix Matrix::cublas_multiply(const Matrix& other) const {
    if (cols != other.rows) {
        throw std::runtime_error("Dimensiones incompatibles para multiplicación cuBLAS");
    }
    
    // Crear handle cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    size_t bytes_a = rows * cols * sizeof(float);
    size_t bytes_b = other.rows * other.cols * sizeof(float);
    size_t bytes_c = rows * other.cols * sizeof(float);
    
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_b, bytes_b);
    hipMalloc(&d_c, bytes_c);
    
    hipMemcpy(d_a, data.data(), bytes_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, other.data.data(), bytes_b, hipMemcpyHostToDevice);
    
    // Parámetros para SGEMM
    const float alpha = 1.0f, beta = 0.0f;
    
    // cuBLAS usa column-major, necesitamos ajustar
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                other.cols, rows, cols,
                &alpha,
                d_b, other.cols,
                d_a, cols,
                &beta,
                d_c, other.cols);
    
    Matrix result(rows, other.cols);
    hipMemcpy(result.data.data(), d_c, bytes_c, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);
    
    return result;
}